#include "hip/hip_runtime.h"

#include "Skybox.hpp"

#include <dirent.h>
#include <sys/stat.h>
#include <sys/types.h>


// a fudge parameter because I am lazy
#ifndef SKYCUBE_FILETYPE
#define SKYCUBE_FILETYPE ".jpg" 
#endif

#define IMG_INDEX(i,j,k, width, height, channels) ((j)*(width)*(channels)+(i)*(channels)+(k))


Skybox::Skybox(const char* filename, bool is_cube, float offset_phi, float offset_th, bool cuda_speedup){
    __is_cube = is_cube;
    __cuda_speedup = cuda_speedup;
    __offset_phi = offset_phi;
    __offset_th = offset_th;

    if(is_cube){
        __img = NULL;

        // its assumed they all have the same dimensions
        // if this is not true, then its all fucked
        // __img_up = stbi_load((String(filename) + "/top" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);
        // __img_down = stbi_load((String(filename) + "/bottom" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);
        // __img_front = stbi_load((String(filename) + "/front" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);
        // __img_back = stbi_load((String(filename) + "/back" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);
        // __img_left = stbi_load((String(filename) + "/left" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);
        // __img_right = stbi_load((String(filename) + "/right" + SKYCUBE_FILETYPE), &__width, &__height, &__channels, 0);

        this->initialize_image((String(filename) + "/top" + SKYCUBE_FILETYPE), __img_up, __d_img_up);
        this->initialize_image((String(filename) + "/bottom" + SKYCUBE_FILETYPE), __img_down, __d_img_down);
        this->initialize_image((String(filename) + "/front" + SKYCUBE_FILETYPE), __img_front, __d_img_front);
        this->initialize_image((String(filename) + "/back" + SKYCUBE_FILETYPE), __img_back, __d_img_back);
        this->initialize_image((String(filename) + "/left" + SKYCUBE_FILETYPE), __img_left, __d_img_left);
        this->initialize_image((String(filename) + "/right" + SKYCUBE_FILETYPE), __img_right, __d_img_right);

        // if(__img_up == NULL 
        //     || __img_down == NULL 
        //     || __img_front == NULL 
        //     || __img_back == NULL
        //     || __img_left == NULL 
        //     || __img_right == NULL){
        //     printf("Error loading some skycube image\n");
        //     exit(1);
        // }

        // if(__cuda_speedup){
        //     hipMalloc((void**) &__d_img_up, __width*__height*__channels*sizeof(uint8_t));
        //     hipMemcpy(__d_img_up, __img_up, __width*__height*__channels*sizeof(uint8_t), hipMemcpyHostToDevice);


        // }


    } else {
        __img_up = NULL;
        __img_down = NULL;
        __img_front = NULL;
        __img_back = NULL;
        __img_left = NULL;
        __img_right = NULL;

        // __img = stbi_load(filename, &__width, &__height, &__channels, 0);

        this->initialize_image(filename, __img, __d_img);

        // if(__img == NULL){
        //     printf("Error loading image\n");
        //     exit(1);
        // }

        // if(__cuda_speedup){
        //     hipMalloc((void**) &__d_img, __width*__height*__channels*sizeof(uint8_t));
        //     hipMemcpy(__d_img, __img, __width*__height*__channels*sizeof(uint8_t), hipMemcpyHostToDevice);
        // }

    }

}

void Skybox::initialize_image(const char* filename, uint8_t* img, uint8_t* d_img){
    img = stbi_load(filename, &__width, &__height, &__channels, 0);

    if(img == NULL){
        printf("Error loading image: %s\n", filename);
        exit(1);
    }

    if(__cuda_speedup){
        hipMalloc((void**) &d_img, __width*__height*__channels*sizeof(uint8_t));
        hipMemcpy(d_img, img, __width*__height*__channels*sizeof(uint8_t), hipMemcpyHostToDevice);
    }
}

Skybox::~Skybox(){

    // Program may die in this distructor, remove free if that happens
    if(__img) free(__img);
    if(__img_up) free(__img_up);
    if(__img_down) free(__img_down);
    if(__img_front) free(__img_front);
    if(__img_left) free(__img_left);
    if(__img_right) free(__img_right);

    if(__d_img) hipFree(__d_img);
    if(__d_img_up) hipFree(__d_img_up);
    if(__d_img_down) hipFree(__d_img_down);
    if(__d_img_front) hipFree(__d_img_front);
    if(__d_img_back) hipFree(__d_img_back);
    if(__d_img_left) hipFree(__d_img_left);
    if(__d_img_right) hipFree(__d_img_right);
}


// phi, th rotstion values for each face
// operation: rotate face back to front
__constant__ float rot_F[2]{0,0};
__constant__ float rot_B[2]{-PI, 0};

__constant__ float rot_R[2]{PI/2, 0};
__constant__ float rot_L[2]{-PI/2, 0};

__constant__ float rot_U[2]{0, PI/2};
__constant__ float rot_D[2]{0, -PI/2};

// __constant__ float 


// phi is along the width, x; th is along height, y
void Skybox::get_pixel(float phi, float th, uint8_t* pixel){
    
    float angle_phi = interval_mod(phi + __offset_phi, -PI, PI);
    float angle_th = interval_mod(th + __offset_th, 0, PI);

    if(__is_cube){
        // TODO: needs some algebra and rays inside a cube
        // referential at center of cube
        // x points FRONT
        // y points LEFT
        // z points UP


        // x = r cos phi sin th
        // y = r sin phi sin th
        // z = r cos th

        // faces: F, B, R, L, U, D
        // front, back, left, right, up, down
        // F: x = 1, y,z<1
        // B: x=-1, y,z<1
        // R (L): y=(-)1, x,z<1 
        // U (D): z=(-)1, z,y<1

        // xr = x/r, yr = y/r, zr = z/r
        float xr = cos(angle_phi) * sin(angle_th);
        float yr = sin(angle_phi) * sin(angle_th);
        float zr = cos(angle_th);

        // figure out if ray pointing to up face
        // assume z = +-1 -> r = +-1/(zr)
        // x = xr*r = +- xr/zr
        // y = yr * r = +- yr/zr 
        
        // if |xr/zr| < 1 and |yr/zr| < 1 
        // then we are in up or down face
        // up face : 0<th<PI/2
        // down face: PI/2<th<0

        // otherwise
        float transformed_phi = angle_phi;
        float transformed_th = angle_th;
        bool back_face = false;
        uint8_t* __img__;
        if(fabs(xr/zr)<1 & fabs(yr/zr)<1){
            // we are in UP or DOWN face

            if(angle_th<PI/2){ // UP
                transformed_phi = angle_phi + rot_U[0];
                transformed_th = angle_th + rot_U[1];
                __img__ = __img_up;
            } else { // DOWN
                transformed_phi = angle_phi + rot_D[0];
                transformed_th = angle_th + rot_D[1];
                __img__ = __img_down;
            }
        } else if(fabs(angle_phi) < PI/4) { // FRONT
            transformed_phi = angle_phi + rot_F[0];
            transformed_th = angle_th + rot_F[1];
            __img__ = __img_front;
        } else if(fabs(angle_phi-PI/2) < PI/4){ // RIGHT
            transformed_phi = angle_phi + rot_R[0];
            transformed_th = angle_th + rot_R[1];
            __img__ = __img_right;
        } else if(fabs(angle_phi+PI/2) < PI/4) { // LEFT
            transformed_phi = angle_phi + rot_R[0];
            transformed_th = angle_th + rot_R[1];
            __img__ = __img_left
        } else { // BACK
            transformed_phi = angle_phi + rot_B[0];
            transformed_th = angle_th + rot_B[1];
            back_face = true;
            __img__ = __img_back;
        }
        // y = r sin phi sin th
        // z = r cos th
        // cube within -1,1 in all axis
        // face projection onto y-z plane
        float tr_y = sin(transformed_phi) * sin(transformed_th);
        float tr_z = cos(transformed_th);

        // (tr_y + 1)/2 is between 0 and 1
        // multiplied by __width gives the pixel position

        // (1-2*back_face) flips the sign when back_face is true
        // this, with -__width*back_face in both indexes is intended to rotate the image 180 degrees
        int i = (1-2*back_face)*(-__width * back_face + (int) round( (tr_y+1)/2)*__width);
        int j = (1-2*back_face)*(-__height * back_face + (int) round( (tr_z+1)/2)*__height);
        // ASSUMPTION: all faces of the cube have the same __width, __height and __channels
        for(int k=0; k<__channels; ++k){
            pixel[k] = __img__[IMG_INDEX(i,j,k, __width, __height, __channels)];
        }

    } else {
        __img__ = __img;
        int i = (int)round( interval_mod(angle_phi, 0, 2*PI)* width/(2*PI)) ;
        int j = (int)round(interval_mod(angle_th, 0, PI) * height/PI);

        for(int k=0; k<__channels; k++){
            pixel[k] = __img[IMG_INDEX(i, j, k, __width, __height, __channels)];
        }
    }


}


