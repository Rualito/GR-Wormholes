#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <SFML/Graphics.hpp>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "math.h"
#define PI 3.1415926535897932384626433832795028841971693993751058209749445923078

#include "limits.h"

#include <iostream>
#include <fstream>

// THIS WORKS, DON'T TWEAK THIS
#define IMG_INDEX(i,j,k, width, height, channels) ((j)*(width)*(channels)+(i)*(channels)+(k))
// channels should always be 4
#define TEXTURE_INDEX(i,j,k, width, height, channels) ((j)*(width)*(channels)+(i)*(channels)+(k))

#define THREADS_PER_BLOCK 16 // 1 dimensional thread blocks
#define THREADS(N) dim3((THREADS_PER_BLOCK<N?THREADS_PER_BLOCK:N),1,1);
#define BLOCKS(N) dim3((N>THREADS_PER_BLOCK?N/THREADS_PER_BLOCK:1),1,1);


// #define BENCHMARK
#include "BenchMark.hpp"

// #define DEBUG
// #define COUNT
#include "Debug.hpp"

#include "cuErrorChecking.cuh"
#include "RK-kernels.cuh"

#define DEBUG_ODE

#define MAX_ITER 2048


typedef bool (*ODEfunc)(int n_param, float* params, int n_coords, float* coords, float* output);

__device__
bool ode_test(int n_params, float *params, int n_coords, float* coords, float *output){
    if (n_params<1 | n_coords<2){
        return false;
    }

    output[0] = 1; // time step iteration
    output[1] = coords[2]; //dx/dt = y 
    output[2] = -params[0]*params[0] * coords[1]; //  dy/dt = -w^2 * x
    // these are equations for the harmonic oscillator

    return true;
}

__device__ ODEfunc ode_test_f = ode_test;

// extern void run_DOPRI5_until(int n_param, float* params, int n_coords, float *coords0, ODEfunc f,
// int MAX_ITER, float* max_coords, int* conv_iter_n, float* coords_iterations, float step_size, uint8_t* flag){

int main(int argc, char** argv){
    // first: test DOPRI5 algorithm - harmonic oscillator
    //  dx/dt = y 
    //  dy/dt = -w^2 * x
    // only one parameter w, but many possible starting conditions

    int n_points = 1024;
    // time, x, y
    float pointsS[3]{0,0.,1.};
    float pointsF[3]{0,1.,0.}; 
    printf("Beginning\n");

    // memory on graphics card
    float *params = new float[1];
    *params = 2*PI;
    float *d_params; // device parameters
    hipMalloc( (void**) &d_params, 1*sizeof(float));
    hipMemcpy(d_params, params, 1*sizeof(float), hipMemcpyHostToDevice);
    printf("params done\n");
    float *d_coordsS; // device coordinates
    hipMalloc( (void**) &d_coordsS, 3*sizeof(float));
    hipMemcpy(d_coordsS, pointsS, 3*sizeof(float), hipMemcpyHostToDevice);

    float *d_coordsF; 
    hipMalloc( (void**) &d_coordsF, 3*sizeof(float));
    hipMemcpy(d_coordsF, pointsF, 3*sizeof(float), hipMemcpyHostToDevice);
    printf("cordsF done\n");

    float *integration_results = (float*) malloc(n_points*MAX_ITER*3*sizeof(float));
    float *d_integration_results;
    hipMalloc( (void**) &d_integration_results, n_points*MAX_ITER*3*sizeof(float));

    float *integration_end = (float*) malloc(n_points*sizeof(float));
    int *d_integration_end;
    hipMalloc( (void**) &d_integration_end, n_points*sizeof(int));
    printf("integration_... done\n");

    float *max_coords = (float*) malloc(3*sizeof(float));
    max_coords[0] = 10; // 10 time units of integration
    max_coords[1] = std::numeric_limits<float>::max();
    max_coords[2] = std::numeric_limits<float>::max();
    
    float *d_max_coords;
    hipMalloc((void**) &d_max_coords, 3*sizeof(float));
    hipMemcpy(d_max_coords, max_coords, 3*sizeof(float), hipMemcpyHostToDevice);
    printf("max_coords done\n");
    uint8_t *flag = new uint8_t[1];
    *flag = 0;
    uint8_t *d_flag;
    hipMalloc((void**) &d_flag, sizeof(uint8_t));
    hipMemcpy(d_flag, flag, sizeof(uint8_t), hipMemcpyHostToDevice);

    // hipMemcpy(d_coords0, coords0, 2*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_params, params, 1*sizeof(float), hipMemcpyHostToDevice);

    ODEfunc *h_ode_test_f;
    hipMemcpyFromSymbol(h_ode_test_f, HIP_SYMBOL(ode_test_f), sizeof(ODEfunc));

    ODEfunc *d_ode_test_f;
    hipMalloc( (void**)&d_ode_test_f, sizeof(ODEfunc));
    hipMemcpy(d_ode_test_f, h_ode_test_f, sizeof(ODEfunc), hipMemcpyHostToDevice);


    float initial_step_size = 1.0e-4f;
    float rtol = 1.0e-6;
    
    dim3 blocks = BLOCKS(n_points);
    dim3 threads = THREADS(n_points);
    printf("Before CUDA\n");
    BENCHMARK_START(0);
    // https://stackoverflow.com/questions/49946929/pass-function-as-parameter-in-cuda-with-static-pointers
    //extern void run_DOPRI5_coord0_range_until(int n_param, float* params, int n_coords, float *coordsS, float *coordsF, int n_points, ODEfunc f,
    //   int MAX_ITER, float* max_coords, int* conv_iter_n, float* coords_iterations_range, float step_size, uint8_t* flag) 
    run_DOPRI5_coord0_range_until<<<blocks, threads>>>(1, d_params, 3, d_coordsS, d_coordsF, n_points, d_ode_test_f, 
        MAX_ITER, d_max_coords, d_integration_end, d_integration_results, initial_step_size, rtol,d_flag);
    
    // can do for loop, change d_params and iterate
    // *params = 2;
    // hipMemcpy(d_params, params, 1*sizeof(float), hipMemcpyHostToDevice);

    gpuErrchk(hipGetLastError(), false);
    hipDeviceSynchronize();
    
    BENCHMARK_END(0);

    // obtaining the results from the device
    hipMemcpy(integration_results, d_integration_results, n_points*MAX_ITER*3*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(integration_end, d_integration_end, n_points*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(flag, d_flag, sizeof(uint8_t), hipMemcpyDeviceToHost);
    
    printf("After CUDA\n");

    printf("Result flag: %d\n", flag);

    std::ofstream outdata;
    outdata.open("integration_results.dat");
    if(!outdata){
        printf("Can't open file to write results\n");
        exit(1);
    }
    float coord0[3];
    for(int i=0; i<n_points; ++i){ // for each requested point
        outdata << "New point coordinate " << i << "\n";
        for(int k=0; k<3; ++k){
            float coord_now = pointsS[k] + ((pointsF[k]-pointsF[k]) * i*1.0) / n_points;
            outdata << coord_now << " ";
        } 
        outdata <<  "\n";

        for(int j=0; j<integration_end[i]; ++j){ // throughout the simulation results
            for(int k=0; k<3; ++k){
                outdata << integration_results[3*MAX_ITER*i + j*3+k] << " ";
                // 3 coordinates
            }
            outdata << "\n";
        }
        outdata << "\n";
    }

    //    T O  D O
    // saving the results to a file, to be later intrepreted by some python code (easier to draw)
        

    hipFree(d_coordsS);
    hipFree(d_coordsF);
    hipFree(d_integration_results);
    hipFree(d_integration_end);
    hipFree(d_ode_test_f);
    hipFree(d_max_coords);
    hipFree(d_flag);

    delete params;
    delete flag;
    free(integration_results);
    free(integration_end);
    free(max_coords);
    
    return 0;
}